#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024
#define tol 0.000001f
#define itr_tol 100

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    out[i] = a[i] + 2.00*b[i];
    //out[i] = a[i] + 2.00*b[i] + out[i];
    //out[i] = a[i] + 2.00*b[i] + 2.00*out[i];
    
}


__global__ void get_residual(float *r, float *a, float *b,float *x, int n) {
    // r = b - Ax
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int ij= 0;
    if(i >= n){
        return;
    }

    for(int j = 0; j < n; j++){
        ij = n*i + j;
        //printf("ij=%i\n",ij);
        r[i] += - a[ij]*x[j];
        //printf("r[%i] = %f, a[i]=%f,b[i]=%f,x[i]=%f \n",i,r[i],a[ij],b[i],x[j]);
    }
    r[i] += b[i];
    //printf("%i, %i, %i, %i, %i,%i\n",blockDim.x, blockDim.y, blockDim.z,gridDim.x,gridDim.y,gridDim.z);
    //printf("%i, %i, %i, %i, %i,%i\n",blockIdx.x, threadIdx.x,i);
    
}


__global__ void cuda_matmul(float *r, float *a,float *x, int n) {
    // r = b - Ax
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int ij= 0;
    if(i >= n){
        return;
    }

    for(int j = 0; j < n; j++){
        ij = n*i + j;
        //printf("ij=%i\n",ij);
        r[i] += a[ij]*x[j];
    }
    //printf("%i, %i, %i, %i, %i,%i\n",blockDim.x, blockDim.y, blockDim.z,gridDim.x,gridDim.y,gridDim.z);
    //printf("%i, %i, %i, %i, %i,%i\n",blockIdx.x, threadIdx.x,i);
    
}


__global__ void cuda_dot_product(float *r1, float *r2, float *rr,int n) {
    // r = b - Ax
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j;
    
    rr[i]=0.0f;

    for(j=0; j<n; j++){
        rr[i] += r1[j]*r2[j];
    }
}


__global__ void cuda_e_residual_bicg(float *e, float *r, float *alp, float *y,int n) {
    // r = b - Ax
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    
    e[i] = r[i] - alp[i]*y[i];
    
}

__global__ void cuda_add_x_alp_p_c3_e(float *x, float *alp, float *p, float *c3, float *e, int n) {
    // r = b - Ax
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    x[i] = x[i] + alp[i]*p[i] + c3[i]*e[i];
    
}


__global__ void cuda_update_p_bicg(float *p, float *r, float *bet, float *c3, float *y, int n) {
    // r = b - Ax
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    p[i] = r[i] + bet[i]*(p[i] - c3[i]*y[i]);
    
}

__global__ void cuda_divide_a_by_b(float *ret, float *a, float *b,int n) {
    // r = b - Ax
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    ret[i] = a[i]/b[i];

}

__global__ void cuda_divide_a_by_b_by_c(float *ret, float *a, float *b,float *c,int n) {
    // r = b - Ax
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    ret[i] = a[i]/b[i]/c[i];

}


__global__ void cuda_check_float(float *x){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    printf("x[%i] : %f\n", i,x[i]);
}


__global__ void cuda_copy_vector(float *x, float *copy){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    copy[i] = x[i];
}



int main(){
    float *a, *b, *x, *out;
    float *r, *r0, *p, *y, *e, *v, *er,*er0;
    float *alp, *bet, *c1,*c2, *c3, *ev, *vv, *rr,*init_rr;

    float *d_a, *d_b, *d_x, *d_out;
    float *d_r, *d_r0, *d_p, *d_y, *d_e, *d_v, *d_er;
    float *d_alp, *d_bet, *d_c1,*d_c2, *d_c3, *d_ev, *d_vv, *d_rr, *d_er0,*d_init_rr;

    // Allocate memory
    a = (float*)malloc(sizeof(float) * N*N);
    b = (float*)malloc(sizeof(float) * N);
    x = (float*)malloc(sizeof(float) * N);
    r = (float*)malloc(sizeof(float) * N);
    r0= (float*)malloc(sizeof(float) * N);
    p = (float*)malloc(sizeof(float) * N);
    y = (float*)malloc(sizeof(float) * N);
    e = (float*)malloc(sizeof(float) * N);
    v = (float*)malloc(sizeof(float) * N);
    
    alp = (float*)malloc(sizeof(float) * N);
    bet = (float*)malloc(sizeof(float) * N);
    c1  = (float*)malloc(sizeof(float) * N);
    c2 = (float*)malloc(sizeof(float) * N);
    c3 = (float*)malloc(sizeof(float) * N);
    ev = (float*)malloc(sizeof(float) * N);
    vv = (float*)malloc(sizeof(float) * N);
    rr= (float*)malloc(sizeof(float) * N);
    er= (float*)malloc(sizeof(float) * N);
    er0= (float*)malloc(sizeof(float) * N);
    init_rr= (float*)malloc(sizeof(float) * N);

    out = (float*)malloc(sizeof(float) * N);
    

    // Initialize array
    int itr;
    itr=0;
    for(int i = 0; i < N*N; i++){
        a[i] = 0.0f;
        if(i==itr*N+itr){
            //if(itr<20){
            //    printf("%i, %i\n",i,itr);    
            //}
            a[i] = 1.0f*float(itr);
            itr = itr + 1;
        }
        
    }
    // a = unit matrix
    
    for(int i = 0; i < N; i++){
        b[i] = 2.0f;
    }

    for(int i = 0; i < N; i++){
        x[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        r[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        r0[i]= 0.0f;
    }
    for(int i = 0; i < N; i++){
        p[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        y[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        e[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        v[i] = 0.0f;
    }

    for(int i = 0; i < N; i++){
        alp[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        bet[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        c1[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        c2[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        c3[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        ev[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        vv[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        rr[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        er[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        er0[i] = 0.0f;
    }
    for(int i = 0; i < N; i++){
        init_rr[i] = 0.0f;
    }
    

    for(int i = 0; i < N; i++){
        out[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N*N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_x, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);
    hipMalloc((void**)&d_r, sizeof(float) * N);
    hipMalloc((void**)&d_r0, sizeof(float) * N);
    hipMalloc((void**)&d_p, sizeof(float) * N);
    hipMalloc((void**)&d_y, sizeof(float) * N);
    hipMalloc((void**)&d_e, sizeof(float) * N);
    hipMalloc((void**)&d_v, sizeof(float) * N);

    hipMalloc((void**)&d_alp, sizeof(float) * N);
    hipMalloc((void**)&d_bet, sizeof(float) * N);
    hipMalloc((void**)&d_c1, sizeof(float) * N);
    hipMalloc((void**)&d_c2, sizeof(float) * N);
    hipMalloc((void**)&d_c3, sizeof(float) * N);
    hipMalloc((void**)&d_ev, sizeof(float) * N);
    hipMalloc((void**)&d_vv, sizeof(float) * N);
    hipMalloc((void**)&d_rr, sizeof(float) * N);
    hipMalloc((void**)&d_er0, sizeof(float) * N);
    hipMalloc((void**)&d_er, sizeof(float) * N);
    hipMalloc((void**)&d_init_rr, sizeof(float) * N);

    
    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_r0, r0, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_p, p, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_e, e, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, sizeof(float) * N, hipMemcpyHostToDevice);

    hipMemcpy(d_alp, alp, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_bet, bet, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_c1, c1, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_c2, c2, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_c3, c3, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_ev, ev, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_vv, vv, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_rr, rr, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_er0, er0, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_init_rr, init_rr, sizeof(float) * N, hipMemcpyHostToDevice);

    // Setting thread
    int blocksize = 512;
    dim3 block (blocksize, 1, 1);
    dim3 grid (N / block.x, 1, 1);




    // Executing kernel for BiCGSTAB
    // a=unit matrix
    // b=2.0
    // x=0.0
    // x=0.0
    get_residual<<<grid, block>>>(d_r, d_a, d_b, d_x, N);
    

    //cudaDeviceSynchronize();
    cuda_dot_product<<<grid, block>>>(d_r,d_r, d_c1, N);
    //r[:] = 2
    //d_c1 = 2*2*1024=4096
    
    //cuda_check_float<<<grid, block>>>(d_c1);
    
    cuda_copy_vector<<<grid, block>>>(d_c1, d_init_rr);

    //c1=init_rr=4096

    //printf("%f",d_c1[0]);
    hipMemcpy(c1, d_c1, sizeof(float) * N, hipMemcpyDeviceToHost);
    
    if(c1[0]>float(tol) ){

        
        cuda_copy_vector<<<grid, block>>>(d_r, d_p);

        cuda_copy_vector<<<grid, block>>>(d_r, d_r0);
        
        for(int k=0; k<itr_tol;k++){
         
            cuda_dot_product<<<grid, block>>>(d_r, d_r0, d_c1, N);
            //d_c1=4*1024=4096

            cuda_matmul<<<grid, block>>>(d_y, d_a, d_p, N);
            //d_y = d_p = 2

            cuda_dot_product<<<grid, block>>>(d_r0, d_y, d_c2, N);
            //d_c2=4*1024=4096

            cuda_divide_a_by_b<<<grid, block>>>(d_alp, d_c1, d_c2, N);
            // alp = 1.0

            cuda_e_residual_bicg<<<grid, block>>>(d_e, d_r, d_alp, d_y, N);

            cuda_matmul<<<grid, block>>>(d_v, d_a, d_e, N);

            cuda_dot_product<<<grid, block>>>(d_e, d_v, d_ev, N);
            cuda_dot_product<<<grid, block>>>(d_v, d_v, d_vv, N);
            
            hipMemcpy(vv, d_vv, sizeof(float) * N, hipMemcpyDeviceToHost);

            if(vv[0]==0.0f ){
                printf("cuda_BiCGSTAB devide by zero\n");
                return 1;
            }else{
                cuda_divide_a_by_b<<<grid, block>>>(d_c3, d_ev, d_vv, N);
                cuda_add_x_alp_p_c3_e<<<grid, block>>>(d_x, d_alp, d_p, d_c3, d_e, N);
                cuda_e_residual_bicg<<<grid, block>>>(d_r, d_e, d_c3, d_v, N); 
                cuda_dot_product<<<grid, block>>>(d_r, d_r, d_rr, N);
                hipMemcpy(rr, d_rr, sizeof(float) * N, hipMemcpyDeviceToHost);
                
                if(rr[0]<tol){
                    break;
                }else{
                    cuda_dot_product<<<grid, block>>>(d_r0, d_r, d_c1, N);
                    cuda_divide_a_by_b_by_c<<<grid, block>>>(d_bet, d_c1, d_c2, d_c3, N);
                    cuda_update_p_bicg<<<grid, block>>>(d_p, d_r, d_bet, d_c3,d_y, N);
                }
            }
            hipMemcpy(rr, d_rr, sizeof(float) * N, hipMemcpyDeviceToHost);
            printf("itr = %i, residual =%f\n",k,rr[0]);
            if(k+1==itr_tol){
                printf("ERROR :: bicgstab did not converge.\n");
                return 1;
            }
           
        }
        

    }





    //vector_add<<<grid,block>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipMemcpy(rr, d_rr, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipMemcpy(x, d_x, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipMemcpy(alp, d_alp, sizeof(float) * N, hipMemcpyDeviceToHost);


    //printf("%f\n", out[0]);
    //printf("alp[0] : %f\n", alp[0]);
    //printf("rr[0] : %f\n", rr[0]);
    //printf("x[0] : %f\n", x[0]);
    
    //printf("%i, %i, %i, %i, %i,%i\n",block.x, block.y, block.z,grid.x,grid.y,grid.z);
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_out);
    
    hipFree(d_r);
    hipFree(d_r0);
    hipFree(d_p);
    hipFree(d_y);
    hipFree(d_e);
    hipFree(d_v);
    hipFree(d_er);
    hipFree(d_er0);
    
    hipFree(d_alp);
    hipFree(d_bet);
    hipFree(d_c1);
    hipFree(d_c2);
    hipFree(d_c3);
    hipFree(d_ev);
    hipFree(d_vv);
    hipFree(d_rr);
    hipFree(d_init_rr);

    
    // Deallocate host memory
    free(a);
    free(b);
    free(x);
    free(out);
    
    free(r);
    free(r0);
    free(p);
    free(y);
    free(e);
    free(v);
    free(er);
    free(er0);
    
    free(alp);
    free(bet);
    free(c1);
    free(c2);
    free(c3);
    free(ev);
    free(vv);
    free(rr);
    free(init_rr);
}
