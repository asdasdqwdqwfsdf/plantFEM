#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 500000000



__global__ void vector_add(float *out, float *a, float *b, int n) {
    int my_number = blockIdx.x+blockIdx.y+threadIdx.x+threadIdx.y+threadIdx.z;
    for(int i = 0; i < n; i++){
        out[i] = a[i] + 2.00*b[i];
        //out[i] = a[i] + 2.00*b[i] + out[i];
        //out[i] = a[i] + 2.00*b[i] + 2.00*out[i];
    }
}



int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    
    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
    
    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    
    // Executing kernel 
    dim3 grid(10,10);
    dim3 block(10,10,10);
    vector_add<<<grid, block>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    printf("%f\n", out[0]);
    
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    
    // Deallocate host memory
    free(a);
    free(b);
    free(out);
}
